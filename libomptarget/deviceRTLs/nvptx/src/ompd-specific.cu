#include "hip/hip_runtime.h"
#include "ompd-specific.h"
#ifdef OMPD_SUPPORT

/**
 * Declaration of symbols to hold struct size and member offset information
 */

__device__ __shared__ static int ompd_target_initialized;
__device__ __shared__ uint64_t ompd_target_access__omptarget_nvptx_TaskDesc__data_items_threadId;
__device__ __shared__ uint64_t ompd_target_sizeof__omptarget_nvptx_TaskDesc__data_items_threadId;

#define ompd_target_declare_access(t,m) __device__ __shared__ uint64_t ompd_target_access__##t##__##m; 
OMPD_FOREACH_ACCESS(ompd_target_declare_access)
#undef ompd_target_declare_access

#define ompd_target_declare_sizeof_member(t,m) __device__ __shared__ uint64_t ompd_target_sizeof__##t##__##m; 
OMPD_FOREACH_ACCESS(ompd_target_declare_sizeof_member)
#undef ompd_target_declare_sizeof_member

#define ompd_target_declare_sizeof(t) __device__ __shared__ uint64_t ompd_target_sizeof__##t; 
OMPD_FOREACH_SIZEOF(ompd_target_declare_sizeof)
#undef ompd_target_declare_sizeof

__device__ void ompd_init ( void )
{
if (ompd_target_initialized)
  return;
  
/**
 * Calculate member offsets for structs and unions
 */
ompd_target_access__omptarget_nvptx_TaskDesc__data_items_threadId = 
          (uint64_t)&(((omptarget_nvptx_TaskDescr*)0)->data.items.threadId);

#define ompd_target_init_access(t,m) ompd_target_access__##t##__##m = (uint64_t)&(((t*)0)->m); 
OMPD_FOREACH_ACCESS(ompd_target_init_access)
#undef ompd_target_init_access

/**
 * Calculate type size information
 */
ompd_target_sizeof__omptarget_nvptx_TaskDesc__data_items_threadId = 
          (uint64_t)sizeof(((omptarget_nvptx_TaskDescr*)0)->data.items.threadId);

#define ompd_target_init_sizeof_member(t,m) ompd_target_sizeof__##t##__##m = sizeof(((t*)0)->m); 
OMPD_FOREACH_ACCESS(ompd_target_init_sizeof_member)
#undef ompd_target_init_sizeof_member

#define ompd_target_init_sizeof(t) ompd_target_sizeof__##t = sizeof(t); 
OMPD_FOREACH_SIZEOF(ompd_target_init_sizeof)
#undef ompd_target_init_sizeof

  ompd_target_initialized = 1;
}

__device__ void ompd_bp_parallel_begin (){ asm (""); }
__device__ void ompd_bp_parallel_end (){ asm (""); }
__device__ void ompd_bp_task_begin (){ asm (""); }
__device__ void ompd_bp_task_end (){ asm (""); }
#endif /* OMPD_SUPPORT */
