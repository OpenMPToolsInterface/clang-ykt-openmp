#include "hip/hip_runtime.h"
#include "ompd-specific.h"
#ifdef OMPD_SUPPORT

/**
 * Declaration of symbols to hold struct size and member offset information
 */

__device__ __shared__ static int ompd_initialized;

#define ompd_declare_access(t,m1,m2,m3) __device__ __shared__ uint64_t ompd_access__##t##__##m1##__##m2##__##m3; 
OMPD_FOREACH_ACCESS3(ompd_declare_access)
#undef ompd_declare_access

#define ompd_declare_access(t,m) __device__ __shared__ uint64_t ompd_access__##t##__##m; 
OMPD_FOREACH_ACCESS(ompd_declare_access)
#undef ompd_declare_access

#define ompd_declare_sizeof_member(t,m) __device__ __shared__ uint64_t ompd_sizeof__##t##__##m; 
OMPD_FOREACH_ACCESS(ompd_declare_sizeof_member)
#undef ompd_declare_sizeof_member

#define ompd_declare_sizeof_member(t,m1,m2,m3) __device__ __shared__ uint64_t ompd_sizeof__##t##__##m1##__##m2##__##m3; 
OMPD_FOREACH_ACCESS3(ompd_declare_sizeof_member)
#undef ompd_declare_sizeof_member

#define ompd_declare_sizeof(t) __device__ __shared__ uint64_t ompd_sizeof__##t; 
OMPD_FOREACH_SIZEOF(ompd_declare_sizeof)
#undef ompd_declare_sizeof

__device__ void ompd_init ( void )
{
if (ompd_initialized)
  return;
  
/**
 * Calculate member offsets for structs and unions
 */
#define ompd_init_access(t,m) ompd_access__##t##__##m = (uint64_t)&(((t*)0)->m); 
OMPD_FOREACH_ACCESS(ompd_init_access)
#undef ompd_init_access

#define ompd_init_access(t,m1,m2,m3) ompd_access__##t##__##m1##__##m2##__##m3 = (uint64_t)&(((t*)0)->m1.m2.m3); 
OMPD_FOREACH_ACCESS3(ompd_init_access)
#undef ompd_init_access

/**
 * Calculate type size information
 */
#define ompd_init_sizeof_member(t,m) ompd_sizeof__##t##__##m = sizeof(((t*)0)->m); 
OMPD_FOREACH_ACCESS(ompd_init_sizeof_member)
#undef ompd_init_sizeof

#define ompd_init_sizeof(t,m1,m2,m3) ompd_sizeof__##t##__##m1##__##m2##__##m3 = sizeof(((t*)0)->m1.m2.m3); 
OMPD_FOREACH_ACCESS3(ompd_init_sizeof)
#undef ompd_init_sizeof

#define ompd_init_sizeof(t) ompd_sizeof__##t = sizeof(t); 
OMPD_FOREACH_SIZEOF(ompd_init_sizeof)
#undef ompd_init_sizeof

  ompd_initialized = 1;
}

__device__ void ompd_bp_parallel_begin (){ asm (""); }
__device__ void ompd_bp_parallel_end (){ asm (""); }
__device__ void ompd_bp_task_begin (){ asm (""); }
__device__ void ompd_bp_task_end (){ asm (""); }
#endif /* OMPD_SUPPORT */
