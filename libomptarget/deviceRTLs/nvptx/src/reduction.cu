#include "hip/hip_runtime.h"
//===---- reduction.cu - NVPTX OpenMP reduction implementation ---- CUDA
//-*-===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
//
// This file contains the implementation of reduction with KMPC interface.
//
//===----------------------------------------------------------------------===//

#include <stdio.h>
#include <complex.h>

#include "../../../deviceRTLs/nvptx/src/omptarget-nvptx.h"

EXTERN void omp_reduction_op_gpu(char *, char *);

// cannot implement atomic_start and atomic_end for GPU. Report runtime error
EXTERN void __kmpc_atomic_start() {
  printf("__kmpc_atomic_start not supported\n");
  asm("trap;");
  return;
}

EXTERN void __kmpc_atomic_end() {
  printf("__kmpc_atomic_end not supported\n");
  asm("trap;");
  return;
}

EXTERN
int32_t __gpu_block_reduce() {
  if (omp_get_num_threads() != blockDim.x)
    return 0;
  unsigned tnum = __ballot(1);
  if (tnum != (~0x0)) { // assume swapSize is 32
    return 0;
  }
  return 1;
}

EXTERN
int32_t __kmpc_reduce_gpu(kmp_Indent *loc, int32_t global_tid, int32_t num_vars,
                          size_t reduce_size, void *reduce_data,
                          void *reduce_array_size, kmp_ReductFctPtr *reductFct,
                          kmp_CriticalName *lck) {
  int globalThreadId = GetGlobalThreadId();
  omptarget_nvptx_TaskDescr *currTaskDescr =
      getMyTopTaskDescriptor(globalThreadId);
  int numthread;
  if (currTaskDescr->IsParallelConstruct()) {
    numthread = omp_get_num_threads();
  } else {
    numthread = omp_get_num_teams();
  }

  if (numthread == 1)
    return 1;
  else if (!__gpu_block_reduce())
    return 2;
  else {
    if (threadIdx.x == 0)
      return 1;
    else
      return 0;
  }
}

EXTERN
int32_t __kmpc_reduce_combined(kmp_Indent *loc) {
  if (threadIdx.x == 0) {
    return 2;
  } else {
    return 0;
  }
}

EXTERN
int32_t __kmpc_reduce41(kmp_Indent *loc, int32_t global_tid, int32_t num_vars,
                        size_t reduce_size, void *reduce_data,
                        void *reduce_array_size, kmp_ReductFctPtr *reductFct,
                        kmp_CriticalName *lck) {
  return __kmpc_reduce_gpu(loc, global_tid, num_vars, reduce_size, reduce_data,
                           reduce_array_size, reductFct, lck);
}

EXTERN
int32_t __kmpc_reduce_nowait41(kmp_Indent *loc, int32_t global_tid,
                               int32_t num_vars, size_t reduce_size,
                               void *reduce_data, void *reduce_array_size,
                               kmp_ReductFctPtr *reductFct,
                               kmp_CriticalName *lck) {
  int globalThreadId = GetGlobalThreadId();
  omptarget_nvptx_TaskDescr *currTaskDescr =
      getMyTopTaskDescriptor(globalThreadId);
  int numthread;
  if (currTaskDescr->IsParallelConstruct()) {
    numthread = omp_get_num_threads();
  } else {
    numthread = omp_get_num_teams();
  }

  if (numthread == 1)
    return 1;
  else if (!__gpu_block_reduce())
    return 2;
  else {
    if (threadIdx.x == 0)
      return 1;
    else
      return 0;
  }
}

EXTERN
void __kmpc_end_reduce(kmp_Indent *loc, int32_t global_tid,
                       kmp_CriticalName *lck) {}

EXTERN
void __kmpc_end_reduce_nowait(kmp_Indent *loc, int32_t global_tid,
                              kmp_CriticalName *lck) {}

// implement different data type or operations  with atomicCAS
#define omptarget_nvptx_add(x, y) ((x) + (y))
#define omptarget_nvptx_sub(x, y) ((x) - (y))
#define omptarget_nvptx_sub_rev(y, x) ((x) - (y))
#define omptarget_nvptx_mul(x, y) ((x) * (y))
#define omptarget_nvptx_div(x, y) ((x) / (y))
#define omptarget_nvptx_div_rev(y, x) ((x) / (y))
#define omptarget_nvptx_min(x, y) ((x) > (y) ? (y) : (x))
#define omptarget_nvptx_max(x, y) ((x) < (y) ? (y) : (x))
#define omptarget_nvptx_andb(x, y) ((x) & (y))
#define omptarget_nvptx_orb(x, y) ((x) | (y))
#define omptarget_nvptx_xor(x, y) ((x) ^ (y))
#define omptarget_nvptx_shl(x, y) ((x) << (y))
#define omptarget_nvptx_shr(x, y) ((x) >> (y))
#define omptarget_nvptx_andl(x, y) ((x) && (y))
#define omptarget_nvptx_orl(x, y) ((x) || (y))
#define omptarget_nvptx_eqv(x, y) ((x) == (y))
#define omptarget_nvptx_neqv(x, y) ((x) != (y))

INLINE __device__ float atomicCAS(float *_addr, float _compare, float _val) {
  int *addr = (int *)_addr;
  int compare = __float_as_int(_compare);
  int val = __float_as_int(_val);
  return __int_as_float(atomicCAS(addr, compare, val));
}

INLINE __device__ double atomicCAS(double *_addr, double _compare,
                                   double _val) {
  unsigned long long int *addr = (unsigned long long int *)_addr;
  unsigned long long int compare = __double_as_longlong(_compare);
  unsigned long long int val = __double_as_longlong(_val);
  return __longlong_as_double(atomicCAS(addr, compare, val));
}

INLINE __device__ long long int
atomicCAS(long long int *_addr, long long int _compare, long long int _val) {
  unsigned long long int *addr = (unsigned long long int *)_addr;
  unsigned long long int compare = (unsigned long long int)(_compare);
  unsigned long long int val = (unsigned long long int)(_val);
  return (long long int)(atomicCAS(addr, compare, val));
}

INLINE __device__ int64_t atomicCAS(int64_t *_addr, int64_t _compare,
                                    int64_t _val) {
  unsigned long long int *addr = (unsigned long long int *)_addr;
  unsigned long long int compare = (unsigned long long int)(_compare);
  unsigned long long int val = (unsigned long long int)(_val);
  return (int64_t)(atomicCAS(addr, compare, val));
}

INLINE __device__ uint64_t atomicCAS(uint64_t *_addr, uint64_t _compare,
                                     uint64_t _val) {
  unsigned long long int *addr = (unsigned long long int *)_addr;
  unsigned long long int compare = (unsigned long long int)(_compare);
  unsigned long long int val = (unsigned long long int)(_val);
  return (uint64_t)(atomicCAS(addr, compare, val));
}

INLINE __device__ float complex atomicCAS(float complex *_addr,
                                          float complex _compare,
                                          float complex _val) {
  double *addr = (double *)_addr;
  double compare = (double)(_compare);
  double val = (double)(_val);
  return (float complex)(atomicCAS(addr, compare, val));
}

#define ATOMIC_GENOP_NATIVE(_name, _dtype, _op, _cudaop)                       \
  EXTERN void __kmpc_atomic_##_name##_##_op(kmp_Indent *id_ref, int32_t gtid,  \
                                            _dtype *lhs, _dtype rhs) {         \
    PRINT(LD_LOOP, "Reduction: thead %d\n", gtid);                             \
    atomic##_cudaop(lhs, rhs);                                                 \
  }                                                                            \
                                                                               \
  EXTERN _dtype __kmpc_atomic_##_name##_##_op##_cpt(                           \
      kmp_Indent *id_ref, int32_t gtid, _dtype *lhs, _dtype rhs, int flag) {   \
    _dtype old = atomic##_cudaop(lhs, rhs);                                    \
    if (flag) {                                                                \
      return omptarget_nvptx_##_op(old, rhs);                                  \
    } else {                                                                   \
      return old;                                                              \
    }                                                                          \
  }

// for types that are supported directly by atomicCAS
#define ATOMIC_GENOP_DIRECT(_name, _dtype, _op)                                \
  EXTERN void __kmpc_atomic_##_name##_##_op(kmp_Indent *id_ref, int32_t gtid,  \
                                            _dtype *lhs, _dtype rhs) {         \
    PRINT(LD_LOOP, "Reduction: thead %d\n", gtid);                             \
    _dtype *temp_lhs = lhs;                                                    \
    _dtype oldvalue = *temp_lhs;                                               \
    _dtype saved;                                                              \
    _dtype newvalue;                                                           \
    do {                                                                       \
      saved = oldvalue;                                                        \
      newvalue = (_dtype)omptarget_nvptx_##_op(saved, rhs);                    \
      oldvalue = atomicCAS(temp_lhs, saved, newvalue);                         \
    } while (saved != oldvalue);                                               \
  }                                                                            \
                                                                               \
  EXTERN _dtype __kmpc_atomic_##_name##_##_op##_cpt(                           \
      kmp_Indent *id_ref, int32_t gtid, _dtype *lhs, _dtype rhs, int flag) {   \
    _dtype *temp_lhs = lhs;                                                    \
    _dtype oldvalue = *temp_lhs;                                               \
    _dtype saved;                                                              \
    _dtype newvalue;                                                           \
    do {                                                                       \
      saved = oldvalue;                                                        \
      newvalue = (_dtype)omptarget_nvptx_##_op(saved, rhs);                    \
      oldvalue = atomicCAS(temp_lhs, saved, newvalue);                         \
    } while (saved != oldvalue);                                               \
    if (flag)                                                                  \
      return newvalue;                                                         \
    else                                                                       \
      return oldvalue;                                                         \
  }

#define ATOMIC_GENOP_DIRECT_REV(_name, _dtype, _op)                            \
  EXTERN void __kmpc_atomic_##_name##_##_op##_rev(                             \
      kmp_Indent *id_ref, int32_t gtid, _dtype *lhs, _dtype rhs) {             \
    _dtype *temp_lhs = lhs;                                                    \
    _dtype oldvalue = *temp_lhs;                                               \
    _dtype saved;                                                              \
    _dtype newvalue;                                                           \
    do {                                                                       \
      saved = oldvalue;                                                        \
      newvalue = (_dtype)omptarget_nvptx_##_op(rhs, saved);                    \
      oldvalue = atomicCAS(temp_lhs, saved, newvalue);                         \
    } while (saved != oldvalue);                                               \
  }                                                                            \
                                                                               \
  EXTERN _dtype __kmpc_atomic_##_name##_##_op##_cpt##_rev(                     \
      kmp_Indent *id_ref, int32_t gtid, _dtype *lhs, _dtype rhs, int flag) {   \
    _dtype *temp_lhs = lhs;                                                    \
    _dtype oldvalue = *temp_lhs;                                               \
    _dtype saved;                                                              \
    _dtype newvalue;                                                           \
    do {                                                                       \
      saved = oldvalue;                                                        \
      newvalue = (_dtype)omptarget_nvptx_##_op(rhs, saved);                    \
      oldvalue = atomicCAS(temp_lhs, saved, newvalue);                         \
    } while (saved != oldvalue);                                               \
    if (flag)                                                                  \
      return newvalue;                                                         \
    else                                                                       \
      return oldvalue;                                                         \
  }

INLINE __device__ void dc_add(double complex *lhs, double complex rhs) {
  double *ptrl = (double *)lhs;
  double *ptrr = (double *)&rhs;
  ptrl[0] += ptrr[0];
  ptrl[1] += ptrr[1];
}

INLINE __device__ void dc_sub(double complex *lhs, double complex rhs) {
  double *ptrl = (double *)lhs;
  double *ptrr = (double *)&rhs;
  ptrl[0] -= ptrr[0];
  ptrl[1] -= ptrr[1];
}

INLINE __device__ void dc_mul(double complex *lhs, double complex rhs) {
  double *ptrl = (double *)lhs;
  double *ptrr = (double *)&rhs;
  double r1 = ptrl[0], r2 = ptrr[0];
  double i1 = ptrl[1], i2 = ptrr[1];
  ptrl[0] = r1 * r2 - i1 * i2;
  ptrl[1] = r1 * i2 + r2 * i1;
}

INLINE __device__ void dc_div(double complex *lhs, double complex rhs) {
  double *ptrl = (double *)lhs;
  double *ptrr = (double *)&rhs;
  double r1 = ptrl[0], r2 = ptrr[0];
  double i1 = ptrl[1], i2 = ptrr[1];
  ptrl[0] = (r1 * r2 + i1 * i2) / (r2 * r2 + i2 * i2);
  ptrl[1] = (i1 * r2 - r1 * i2) / (r2 * r2 + i2 * i2);
}

#define ATOMIC_GENOP_DC(_op)                                                   \
  EXTERN void __kmpc_atomic_cmplx8_##_op(kmp_Indent *id_ref, int32_t gtid,     \
                                         double _Complex *lhs,                 \
                                         double _Complex rhs) {                \
    printf("double complex atomic opertion not supported\n");                  \
    asm("trap;");                                                              \
    return;                                                                    \
  }                                                                            \
  EXTERN double _Complex __gpu_warpBlockRedu_cmplx8_##_op(                     \
      double _Complex rhs) {                                                   \
    __shared__ double _Complex lhs;                                            \
    if (threadIdx.x == 0)                                                      \
      lhs = rhs;                                                               \
    __syncthreads();                                                           \
    for (int i = 1; i < blockDim.x; i++) {                                     \
      if (threadIdx.x == i) {                                                  \
        dc_##_op(&lhs, rhs);                                                   \
      }                                                                        \
      __syncthreads();                                                         \
    }                                                                          \
    return lhs;                                                                \
  }

// implementation with shared
#define ATOMIC_GENOP_DC_obsolete(_op)                                          \
  EXTERN void __kmpc_atomic_cmplx16_##_op(kmp_Indent *id_ref, int32_t gtid,    \
                                          double _Complex *lhs,                \
                                          double _Complex rhs) {               \
    __shared__ unsigned int stepinblock;                                       \
    unsigned tnum = __ballot(1);                                               \
    if (tnum != (~0x0)) {                                                      \
      return;                                                                  \
    }                                                                          \
    if (threadIdx.x == 0)                                                      \
      stepinblock = 0;                                                         \
    __syncthreads();                                                           \
    while (stepinblock < blockDim.x) {                                         \
      if (threadIdx.x == stepinblock) {                                        \
        dc_##_op(lhs, rhs);                                                    \
        stepinblock++;                                                         \
      }                                                                        \
      __syncthreads();                                                         \
    }                                                                          \
  }

ATOMIC_GENOP_DC(add);
ATOMIC_GENOP_DC(sub);
ATOMIC_GENOP_DC(mul);
ATOMIC_GENOP_DC(div);

INLINE __device__ uint64_t fc_add(float r1, float i1, float r2, float i2) {
  uint64_t result;
  float *rr = (float *)&result;
  float *ri = rr + 1;
  *rr = r1 + r2;
  *ri = i1 + i2;
  return result;
}

INLINE __device__ uint64_t fc_sub(float r1, float i1, float r2, float i2) {
  uint64_t result;
  float *rr = (float *)&result;
  float *ri = rr + 1;
  *rr = r1 - r2;
  *ri = i1 - i2;
  return result;
}

INLINE __device__ uint64_t fc_mul(float r1, float i1, float r2, float i2) {
  uint64_t result;
  float *rr = (float *)&result;
  float *ri = rr + 1;
  *rr = r1 * r2 - i1 * i2;
  *ri = r1 * i2 + r2 * i1;
  return result;
}

INLINE __device__ uint64_t fc_div(float r1, float i1, float r2, float i2) {
  uint64_t result;
  float *rr = (float *)&result;
  float *ri = rr + 1;
  *rr = (r1 * r2 + i1 * i2) / (r2 * r2 + i2 * i2);
  *ri = (i1 * r2 - r1 * i2) / (r2 * r2 + i2 * i2);
  return result;
}

#define ATOMIC_GENOP_FC(_op)                                                   \
  EXTERN void __kmpc_atomic_cmplx4_##_op(kmp_Indent *id_ref, int32_t gtid,     \
                                         float complex *lhs,                   \
                                         float complex rhs) {                  \
    uint64_t *temp_lhs = (uint64_t *)lhs;                                      \
    uint64_t oldvalue = *temp_lhs;                                             \
    uint64_t saved;                                                            \
    float *pr1 = (float *)&rhs;                                                \
    float *pi1 = pr1 + 1;                                                      \
    float r1 = *pr1;                                                           \
    float i1 = *pi1;                                                           \
    uint64_t newvalue;                                                         \
    do {                                                                       \
      saved = oldvalue;                                                        \
      float *pr2 = (float *)&saved;                                            \
      float *pi2 = pr2 + 1;                                                    \
      newvalue = fc_##_op(*pr2, *pi2, r1, i1);                                 \
      oldvalue = atomicCAS(temp_lhs, saved, newvalue);                         \
    } while (saved != oldvalue);                                               \
  }                                                                            \
                                                                               \
  EXTERN void __kmpc_atomic_cmplx4_##_op##_cpt(                                \
      kmp_Indent *id_ref, int32_t gtid, float complex *lhs, float complex rhs, \
      float complex *outp, int flag) {                                         \
    uint64_t *temp_lhs = (uint64_t *)lhs;                                      \
    uint64_t oldvalue = *temp_lhs;                                             \
    uint64_t saved;                                                            \
    float *pr1 = (float *)&rhs;                                                \
    float *pi1 = pr1 + 1;                                                      \
    float r1 = *pr1;                                                           \
    float i1 = *pi1;                                                           \
    uint64_t newvalue;                                                         \
    do {                                                                       \
      saved = oldvalue;                                                        \
      float *pr2 = (float *)&saved;                                            \
      float *pi2 = pr2 + 1;                                                    \
      newvalue = fc_##_op(*pr2, *pi2, r1, i1);                                 \
      oldvalue = atomicCAS(temp_lhs, saved, newvalue);                         \
    } while (saved != oldvalue);                                               \
    if (flag) {                                                                \
      float complex *temp = (float complex *)&newvalue;                        \
      *outp = *temp;                                                           \
    } else {                                                                   \
      float complex *temp = (float complex *)&saved;                           \
      *outp = *temp;                                                           \
    }                                                                          \
  }

#define ATOMIC_GENOP_FC_REV(_op)                                               \
  EXTERN void __kmpc_atomic_cmplx4_##_op##_rev(                                \
      kmp_Indent *id_ref, int32_t gtid, float complex *lhs,                    \
      float complex rhs) {                                                     \
    uint64_t *temp_lhs = (uint64_t *)lhs;                                      \
    uint64_t oldvalue = *temp_lhs;                                             \
    uint64_t saved;                                                            \
    float *pr1 = (float *)&rhs;                                                \
    float *pi1 = pr1 + 1;                                                      \
    float r1 = *pr1;                                                           \
    float i1 = *pi1;                                                           \
    uint64_t newvalue;                                                         \
    do {                                                                       \
      saved = oldvalue;                                                        \
      float *pr2 = (float *)&saved;                                            \
      float *pi2 = pr2 + 1;                                                    \
      newvalue = fc_##_op(r1, i1, *pr2, *pi2);                                 \
      oldvalue = atomicCAS(temp_lhs, saved, newvalue);                         \
    } while (saved != oldvalue);                                               \
  }                                                                            \
                                                                               \
  EXTERN void __kmpc_atomic_cmplx4_##_op##_cpt##_rev(                          \
      kmp_Indent *id_ref, int32_t gtid, float complex *lhs, float complex rhs, \
      float complex *outp, int flag) {                                         \
    uint64_t *temp_lhs = (uint64_t *)lhs;                                      \
    uint64_t oldvalue = *temp_lhs;                                             \
    uint64_t saved;                                                            \
    float *pr1 = (float *)&rhs;                                                \
    float *pi1 = pr1 + 1;                                                      \
    float r1 = *pr1;                                                           \
    float i1 = *pi1;                                                           \
    uint64_t newvalue;                                                         \
    do {                                                                       \
      saved = oldvalue;                                                        \
      float *pr2 = (float *)&saved;                                            \
      float *pi2 = pr2 + 1;                                                    \
      newvalue = fc_##_op(r1, i1, *pr2, *pi2);                                 \
      oldvalue = atomicCAS(temp_lhs, saved, newvalue);                         \
    } while (saved != oldvalue);                                               \
    if (flag) {                                                                \
      float complex *temp = (float complex *)&newvalue;                        \
      *outp = *temp;                                                           \
    } else {                                                                   \
      float complex *temp = (float complex *)&saved;                           \
      *outp = *temp;                                                           \
    }                                                                          \
  }

ATOMIC_GENOP_FC(add);
ATOMIC_GENOP_FC(sub);
ATOMIC_GENOP_FC_REV(sub);
ATOMIC_GENOP_FC(mul);
ATOMIC_GENOP_FC(div);
ATOMIC_GENOP_FC_REV(div);

// for int and unit
#define ATOMIC_GENOP_ALL_MIXED(_name, _dirname, _tname, _optype)               \
  _dirname(_tname, _optype, add, Add);                                         \
  _dirname(_tname, _optype, sub, Sub);                                         \
  _name##_REV(_tname, _optype, sub);                                           \
  _name(_tname, _optype, mul);                                                 \
  _name(_tname, _optype, div);                                                 \
  _name##_REV(_tname, _optype, div);                                           \
  _dirname(_tname, _optype, min, Min);                                         \
  _dirname(_tname, _optype, max, Max);                                         \
  _dirname(_tname, _optype, andb, And);                                        \
  _dirname(_tname, _optype, orb, Or);                                          \
  _dirname(_tname, _optype, xor, Xor);                                         \
  _name(_tname, _optype, shl);                                                 \
  _name(_tname, _optype, shr);                                                 \
  _name(_tname, _optype, andl);                                                \
  _name(_tname, _optype, orl);                                                 \
  _name(_tname, _optype, eqv);                                                 \
  _name(_tname, _optype, neqv);

#define ATOMIC_GENOP_ALL_MIXED_FIXED8U(_name, _dirname, _tname, _optype)       \
  _dirname(_tname, _optype, add, Add);                                         \
  _name(_tname, _optype, sub);                                                 \
  _name##_REV(_tname, _optype, sub);                                           \
  _name(_tname, _optype, mul);                                                 \
  _name(_tname, _optype, div);                                                 \
  _name##_REV(_tname, _optype, div);                                           \
  _dirname(_tname, _optype, min, Min);                                         \
  _dirname(_tname, _optype, max, Max);                                         \
  _dirname(_tname, _optype, andb, And);                                        \
  _dirname(_tname, _optype, orb, Or);                                          \
  _dirname(_tname, _optype, xor, Xor);                                         \
  _name(_tname, _optype, shl);                                                 \
  _name(_tname, _optype, shr);                                                 \
  _name(_tname, _optype, andl);                                                \
  _name(_tname, _optype, orl);                                                 \
  _name(_tname, _optype, eqv);                                                 \
  _name(_tname, _optype, neqv);

#define ATOMIC_GENOP_ALL(_name, _tname, _optype)                               \
  _name(_tname, _optype, add);                                                 \
  _name(_tname, _optype, sub);                                                 \
  _name##_REV(_tname, _optype, sub);                                           \
  _name(_tname, _optype, mul);                                                 \
  _name(_tname, _optype, div);                                                 \
  _name##_REV(_tname, _optype, div);                                           \
  _name(_tname, _optype, min);                                                 \
  _name(_tname, _optype, max);                                                 \
  _name(_tname, _optype, andb);                                                \
  _name(_tname, _optype, orb);                                                 \
  _name(_tname, _optype, xor);                                                 \
  _name(_tname, _optype, shl);                                                 \
  _name(_tname, _optype, shr);                                                 \
  _name(_tname, _optype, andl);                                                \
  _name(_tname, _optype, orl);                                                 \
  _name(_tname, _optype, eqv);                                                 \
  _name(_tname, _optype, neqv);

#define ATOMIC_GENOP_FLOAT(_name, _tname, _optype)                             \
  _name(_tname, _optype, add);                                                 \
  _name(_tname, _optype, sub);                                                 \
  _name##_REV(_tname, _optype, sub);                                           \
  _name(_tname, _optype, mul);                                                 \
  _name(_tname, _optype, div);                                                 \
  _name##_REV(_tname, _optype, div);                                           \
  _name(_tname, _optype, min);                                                 \
  _name(_tname, _optype, max);

ATOMIC_GENOP_ALL_MIXED(ATOMIC_GENOP_DIRECT, ATOMIC_GENOP_NATIVE, fixed4,
                       int32_t);
ATOMIC_GENOP_ALL_MIXED(ATOMIC_GENOP_DIRECT, ATOMIC_GENOP_NATIVE, fixed4u,
                       uint32_t);
ATOMIC_GENOP_ALL(ATOMIC_GENOP_DIRECT, fixed8, int64_t);
ATOMIC_GENOP_ALL(ATOMIC_GENOP_DIRECT, fixed8u, uint64_t);
ATOMIC_GENOP_FLOAT(ATOMIC_GENOP_DIRECT, float4, float);
ATOMIC_GENOP_FLOAT(ATOMIC_GENOP_DIRECT, float8, double);

//
// data type of size not 32 nor 64
//

typedef enum {
  omptarget_nvptx_inc,
  omptarget_nvptx_dec,
  omptarget_nvptx_add,
  omptarget_nvptx_sub,
  omptarget_nvptx_sub_rev,
  omptarget_nvptx_mul,
  omptarget_nvptx_div,
  omptarget_nvptx_div_rev,
  omptarget_nvptx_min,
  omptarget_nvptx_max,
  omptarget_nvptx_rd,
  omptarget_nvptx_wr,
  omptarget_nvptx_swp,
  omptarget_nvptx_andb,
  omptarget_nvptx_orb,
  omptarget_nvptx_xor,
  omptarget_nvptx_andl,
  omptarget_nvptx_orl,
  omptarget_nvptx_eqv,
  omptarget_nvptx_neqv,
  omptarget_nvptx_shl,
  omptarget_nvptx_shl_rev,
  omptarget_nvptx_shr,
  omptarget_nvptx_shr_rev,
} omptarget_nvptx_BINOP_t;

template <typename OpType,              // type of the operation performed
          omptarget_nvptx_BINOP_t binop // enum describing the operation
          >
INLINE __device__ OpType Compute(OpType a,
                                 OpType b) // a is old value, b is new value
{
  OpType res = 0;
  if (binop == omptarget_nvptx_inc)
    res = a + b;
  if (binop == omptarget_nvptx_dec)
    res = a - b;
  if (binop == omptarget_nvptx_add)
    res = a + b;
  if (binop == omptarget_nvptx_sub)
    res = a - b;
  if (binop == omptarget_nvptx_sub_rev)
    res = b - a;
  if (binop == omptarget_nvptx_mul)
    res = a * b;
  if (binop == omptarget_nvptx_div)
    res = a / b;
  if (binop == omptarget_nvptx_div_rev)
    res = b / a;
  if (binop == omptarget_nvptx_min)
    res = a < b ? a : b;
  if (binop == omptarget_nvptx_max)
    res = a > b ? a : b;
  if (binop == omptarget_nvptx_rd)
    res = a; // read
  if (binop == omptarget_nvptx_wr)
    res = b; // write and swap are the same
  if (binop == omptarget_nvptx_swp)
    res = b; // write and swap are the same
  if (binop == omptarget_nvptx_andb)
    res = a & b;
  if (binop == omptarget_nvptx_orb)
    res = a | b;
  if (binop == omptarget_nvptx_xor)
    res = a ^ b;
  if (binop == omptarget_nvptx_andl)
    res = a && b;
  if (binop == omptarget_nvptx_orl)
    res = a || b;
  if (binop == omptarget_nvptx_eqv)
    res = a == b;
  if (binop == omptarget_nvptx_neqv)
    res = a != b;
  if (binop == omptarget_nvptx_shl)
    res = a << b;
  if (binop == omptarget_nvptx_shl_rev)
    res = b << a;
  if (binop == omptarget_nvptx_shr)
    res = a >> b;
  if (binop == omptarget_nvptx_shr_rev)
    res = b >> a;

  return res;
}

template <>
INLINE __device__ float Compute<float, omptarget_nvptx_add>(float a, float b) {
  return a + b;
}

template <>
INLINE __device__ float Compute<float, omptarget_nvptx_sub>(float a, float b) {
  return a - b;
}

template <>
INLINE __device__ float Compute<float, omptarget_nvptx_mul>(float a, float b) {
  return a * b;
}

template <>
INLINE __device__ float Compute<float, omptarget_nvptx_div>(float a, float b) {
  return a / b;
}

template <>
INLINE __device__ float Compute<float, omptarget_nvptx_min>(float a, float b) {
  return a < b ? a : b;
}

template <>
INLINE __device__ float Compute<float, omptarget_nvptx_max>(float a, float b) {
  return a > b ? a : b;
}

template <>
INLINE __device__ double Compute<double, omptarget_nvptx_add>(double a,
                                                              double b) {
  return a + b;
}

template <>
INLINE __device__ double Compute<double, omptarget_nvptx_sub>(double a,
                                                              double b) {
  return a - b;
}

template <>
INLINE __device__ double Compute<double, omptarget_nvptx_mul>(double a,
                                                              double b) {
  return a * b;
}

template <>
INLINE __device__ double Compute<double, omptarget_nvptx_div>(double a,
                                                              double b) {
  return a / b;
}

template <>
INLINE __device__ double Compute<double, omptarget_nvptx_min>(double a,
                                                              double b) {
  return a < b ? a : b;
}

template <>
INLINE __device__ double Compute<double, omptarget_nvptx_max>(double a,
                                                              double b) {
  return a > b ? a : b;
}

////////////////////////////////////////////////////////////////////////////////
// common atomic slicing functions (modifying only a part of a word)
////////////////////////////////////////////////////////////////////////////////

template <typename MemType, // type of the underlying atomic memory operation
          typename OpType   // type of the operation performed
          >
INLINE __device__ void ComputeAtomic_PrepareSlice(
    OpType *addr,         // original address
    MemType **memAddrPtr, // truncated address to MemType boundary
    MemType
        *memBitShiftRightPtr, // bits to shift to move val to rightmost position
    MemType *memValMaskInPlacePtr) // mask of val in proper position
{
  // compute the mask that corresponds to the natural alignment of memType
  // int -> 0x3; long long -> 0x7
  unsigned long memAddrMask = sizeof(MemType) - 1;
  // compute the addr of the atomic variable truncated to alignment of memType
  *memAddrPtr = (MemType *)((unsigned long)addr & ~memAddrMask);
  // compute the number of bit shift to move the target atomic value in
  // the rightmost position
  unsigned long byteOffsetInMem = (unsigned long)addr & memAddrMask;

  // assumes little-endian
  unsigned long byteShiftRight = byteOffsetInMem;
  *memBitShiftRightPtr = (MemType)(byteShiftRight << 3); // 3: byte to bits

  // mask to isolate target atomic value located in rightmost position
  MemType memValMask = ((MemType)1 << (sizeof(OpType) << 3)) - 1;
  // mask to isolate target atomic value located in place
  *memValMaskInPlacePtr = memValMask << *memBitShiftRightPtr;
}

template <typename MemType, // type of the underlying atomic memory operation
          typename OpType,  // type of the operation performed
          omptarget_nvptx_BINOP_t binop // enum describing the operation
          >
INLINE __device__ MemType ComputeAtomic_ComputeSlice(
    MemType oldMemVal,        // old value
    OpType val,               // value to compute with
    MemType memBitShiftRight, // bits to shift to move val to rightmost position
    MemType memValMaskInPlace // mask of val in proper position
    ) {
  OpType oldValtmp;
  OpType newValtmp;
  // select target atomic val
  MemType oldMemVal_targetVal = oldMemVal & memValMaskInPlace;
  MemType oldMemVal_otherVal = oldMemVal & ~memValMaskInPlace;
  // shift target atomic val to rightmost place: this is the old value

  // type conversion??
  oldValtmp = (OpType)(oldMemVal_targetVal >> memBitShiftRight);
  // perform op

  newValtmp = Compute<OpType, binop>(oldValtmp, val);

  // insert new value in old world mem

  // type conversion??
  MemType newMemVal_targetVal = ((MemType)newValtmp) << memBitShiftRight;
  newMemVal_targetVal &= memValMaskInPlace;
  MemType newMemVal = oldMemVal_otherVal | newMemVal_targetVal;
  return newMemVal;
}

#define ATOMIC_GENOP_PARTIAL(_name, _dtype, _op, _memType)                     \
  EXTERN void __kmpc_atomic_##_name##_##_op(kmp_Indent *id_ref, int32_t gtid,  \
                                            _dtype *lhs, _dtype rhs) {         \
    _memType *memAddr;                                                         \
    _memType memBitShiftRightPtr;                                              \
    _memType memValMaskInPlacePtr;                                             \
    ComputeAtomic_PrepareSlice<_memType, _dtype>(                              \
        lhs, &memAddr, &memBitShiftRightPtr, &memValMaskInPlacePtr);           \
    _memType oldMemVal, newMemVal;                                             \
    oldMemVal = *memAddr;                                                      \
    _memType savedMemVal;                                                      \
    do {                                                                       \
      savedMemVal = oldMemVal;                                                 \
      newMemVal =                                                              \
          ComputeAtomic_ComputeSlice<_memType, _dtype, omptarget_nvptx_##_op>( \
              oldMemVal, rhs, memBitShiftRightPtr, memValMaskInPlacePtr);      \
      oldMemVal = atomicCAS(memAddr, savedMemVal, newMemVal);                  \
    } while (savedMemVal != oldMemVal);                                        \
  }                                                                            \
                                                                               \
  EXTERN _dtype __kmpc_atomic_##_name##_##_op##_cpt(                           \
      kmp_Indent *id_ref, int32_t gtid, _dtype *lhs, _dtype rhs, int flag) {   \
    _memType *memAddr;                                                         \
    _memType memBitShiftRightPtr;                                              \
    _memType memValMaskInPlacePtr;                                             \
    ComputeAtomic_PrepareSlice<_memType, _dtype>(                              \
        lhs, &memAddr, &memBitShiftRightPtr, &memValMaskInPlacePtr);           \
    _memType oldMemVal, newMemVal;                                             \
    oldMemVal = *memAddr;                                                      \
    _memType savedMemVal;                                                      \
    do {                                                                       \
      savedMemVal = oldMemVal;                                                 \
      newMemVal =                                                              \
          ComputeAtomic_ComputeSlice<_memType, _dtype, omptarget_nvptx_##_op>( \
              oldMemVal, rhs, memBitShiftRightPtr, memValMaskInPlacePtr);      \
      oldMemVal = atomicCAS(memAddr, savedMemVal, newMemVal);                  \
    } while (savedMemVal != oldMemVal);                                        \
    if (flag)                                                                  \
      return (_dtype)((newMemVal & memValMaskInPlacePtr) >>                    \
                      memBitShiftRightPtr);                                    \
    else                                                                       \
      return (_dtype)((oldMemVal & memValMaskInPlacePtr) >>                    \
                      memBitShiftRightPtr);                                    \
  }

#define ATOMIC_GENOP_PARTIAL_REV(_name, _dtype, _op, _memType)                 \
  EXTERN void __kmpc_atomic_##_name##_##_op##_rev(                             \
      kmp_Indent *id_ref, int32_t gtid, _dtype *lhs, _dtype rhs) {             \
    _memType *memAddr;                                                         \
    _memType memBitShiftRightPtr;                                              \
    _memType memValMaskInPlacePtr;                                             \
    ComputeAtomic_PrepareSlice<_memType, _dtype>(                              \
        lhs, &memAddr, &memBitShiftRightPtr, &memValMaskInPlacePtr);           \
    _memType oldMemVal, newMemVal;                                             \
    oldMemVal = *memAddr;                                                      \
    _memType savedMemVal;                                                      \
    do {                                                                       \
      savedMemVal = oldMemVal;                                                 \
      newMemVal =                                                              \
          ComputeAtomic_ComputeSlice<_memType, _dtype, omptarget_nvptx_##_op>( \
              oldMemVal, rhs, memBitShiftRightPtr, memValMaskInPlacePtr);      \
      oldMemVal = atomicCAS(memAddr, savedMemVal, newMemVal);                  \
    } while (savedMemVal != oldMemVal);                                        \
  }                                                                            \
                                                                               \
  EXTERN _dtype __kmpc_atomic_##_name##_##_op##_cpt_rev(                       \
      kmp_Indent *id_ref, int32_t gtid, _dtype *lhs, _dtype rhs, int flag) {   \
    _memType *memAddr;                                                         \
    _memType memBitShiftRightPtr;                                              \
    _memType memValMaskInPlacePtr;                                             \
    ComputeAtomic_PrepareSlice<_memType, _dtype>(                              \
        lhs, &memAddr, &memBitShiftRightPtr, &memValMaskInPlacePtr);           \
    _memType oldMemVal, newMemVal;                                             \
    oldMemVal = *memAddr;                                                      \
    _memType savedMemVal;                                                      \
    do {                                                                       \
      savedMemVal = oldMemVal;                                                 \
      newMemVal =                                                              \
          ComputeAtomic_ComputeSlice<_memType, _dtype, omptarget_nvptx_##_op>( \
              oldMemVal, rhs, memBitShiftRightPtr, memValMaskInPlacePtr);      \
      oldMemVal = atomicCAS(memAddr, savedMemVal, newMemVal);                  \
    } while (savedMemVal != oldMemVal);                                        \
    if (flag)                                                                  \
      return (_dtype)((newMemVal & memValMaskInPlacePtr) >>                    \
                      memBitShiftRightPtr);                                    \
    else                                                                       \
      return (_dtype)((oldMemVal & memValMaskInPlacePtr) >>                    \
                      memBitShiftRightPtr);                                    \
  }

#define ATOMIC_GENOP_ALL4(_name, _tname, _optype, _memtype)                    \
  _name(_tname, _optype, add, _memtype);                                       \
  _name(_tname, _optype, sub, _memtype);                                       \
  _name##_REV(_tname, _optype, sub_rev, _memtype);                             \
  _name(_tname, _optype, mul, _memtype);                                       \
  _name(_tname, _optype, div, _memtype);                                       \
  _name##_REV(_tname, _optype, div_rev, _memtype);                             \
  _name(_tname, _optype, min, _memtype);                                       \
  _name(_tname, _optype, max, _memtype);                                       \
  _name(_tname, _optype, andb, _memtype);                                      \
  _name(_tname, _optype, orb, _memtype);                                       \
  _name(_tname, _optype, xor, _memtype);                                       \
  _name(_tname, _optype, andl, _memtype);                                      \
  _name(_tname, _optype, orl, _memtype);                                       \
  _name(_tname, _optype, eqv, _memtype);                                       \
  _name(_tname, _optype, neqv, _memtype);                                      \
  _name(_tname, _optype, shl, _memtype);                                       \
  _name(_tname, _optype, shr, _memtype);

ATOMIC_GENOP_ALL4(ATOMIC_GENOP_PARTIAL, fixed1, int8_t, int32_t);
ATOMIC_GENOP_ALL4(ATOMIC_GENOP_PARTIAL, fixed1u, uint8_t, int32_t);
ATOMIC_GENOP_ALL4(ATOMIC_GENOP_PARTIAL, fixed2u, uint16_t, int32_t);
ATOMIC_GENOP_ALL4(ATOMIC_GENOP_PARTIAL, fixed2, int16_t, int32_t);

// cooperative reduction
// make use of warp, shared variable, and __syncthreads

template <typename T>
INLINE __device__ T myshfldown(T val, unsigned int delta, int size = warpSize) {
  return __shfl_down(val, delta, size);
}

template <>
INLINE __device__ int myshfldown<int>(int val, unsigned int delta, int size) {
  return __shfl_down(val, delta, size);
}

template <>
INLINE __device__ unsigned int
myshfldown<unsigned int>(unsigned int val, unsigned int delta, int size) {
  return __shfl_down(val, delta, size);
}

template <>
INLINE __device__ int64_t myshfldown<int64_t>(int64_t val, unsigned int delta,
                                              int size) {
  return __shfl_down(val, delta, size);
}

template <>
INLINE __device__ uint64_t myshfldown<uint64_t>(uint64_t val,
                                                unsigned int delta, int size) {
  return __shfl_down(val, delta, size);
}

template <>
INLINE __device__ float myshfldown<float>(float val, unsigned int delta,
                                          int size) {
  return __shfl_down(val, delta, size);
}

template <>
INLINE __device__ double myshfldown<double>(double val, unsigned int delta,
                                            int size) {
  return __shfl_down(val, delta, size);
}

template <>
INLINE __device__ unsigned long long
myshfldown<unsigned long long>(unsigned long long val, unsigned int delta,
                               int size) {
  return __shfl_down(val, delta, size);
}

template <typename T, omptarget_nvptx_BINOP_t binop>
__inline__ __device__ T reduInitVal() {
  switch (binop) {
  case omptarget_nvptx_inc:
  case omptarget_nvptx_dec:
  case omptarget_nvptx_add:
  case omptarget_nvptx_sub:
  case omptarget_nvptx_sub_rev:
    return (T)0;
  case omptarget_nvptx_mul:
  case omptarget_nvptx_div:
    return (T)1;
  default:
    return (T)0;
  }
}

template <typename T, omptarget_nvptx_BINOP_t binop>
__inline__ __device__ T warpReduceSum(T val, unsigned int size) {
  for (int offset = size / 2; offset > 0; offset /= 2)
    val = Compute<T, binop>(val, myshfldown<T>(val, offset, size));
  return val;
}

#define MYGSIZE 32

template <typename T, omptarget_nvptx_BINOP_t binop>
__inline__ __device__ T warpBlockReduction(T inputval) {
  __shared__ T shared[MYGSIZE];

  unsigned int remainder = blockDim.x & (MYGSIZE - 1);
  ;
  unsigned int start_r = blockDim.x - remainder;
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  if (blockDim.x < MYGSIZE) {
    shared[threadIdx.x] = inputval;
  } else {
    if (threadIdx.x >= start_r) {
      shared[threadIdx.x - start_r] = inputval;
    } else if (threadIdx.x < MYGSIZE && threadIdx.x >= remainder) {
      shared[threadIdx.x] = reduInitVal<T, binop>();
    }
  }
  __syncthreads();

  if (blockDim.x < MYGSIZE) {
    if (threadIdx.x == 0) {
      T val = shared[0];
      for (unsigned i = 1; i < blockDim.x; i++) {
        val = Compute<T, binop>(val, shared[i]);
      }
      return val;
    }
    return (T)0;
  }

  if (threadIdx.x < start_r) {
    T val = warpReduceSum<T, binop>(inputval, MYGSIZE);
    if (lane == 0) {
      shared[wid] = Compute<T, binop>(shared[wid], val);
    }
  }
  __syncthreads();

  if (wid == 0) {
    T val = warpReduceSum<T, binop>(shared[threadIdx.x], MYGSIZE);
    if (threadIdx.x == 0) {
      return val;
    }
  }
  return (T)0;
}

#define WARPBLOCK_GENREDU(_name, _dtype, _op)                                  \
  EXTERN _dtype __gpu_warpBlockRedu_##_name##_##_op(_dtype rhs) {              \
    return warpBlockReduction<_dtype, omptarget_nvptx_##_op>(rhs);             \
  }

#define WARPBLOCK_GENREDU_ALLOP(_name, _dtype)                                 \
  WARPBLOCK_GENREDU(_name, _dtype, add);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, sub);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, mul);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, div);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, min);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, max);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, andb);                                      \
  WARPBLOCK_GENREDU(_name, _dtype, orb);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, xor);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, andl);                                      \
  WARPBLOCK_GENREDU(_name, _dtype, orl);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, eqv);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, neqv);                                      \
  WARPBLOCK_GENREDU(_name, _dtype, shl);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, shr);

WARPBLOCK_GENREDU_ALLOP(fixed1, int8_t);
WARPBLOCK_GENREDU_ALLOP(fixed1u, uint8_t);
WARPBLOCK_GENREDU_ALLOP(fixed2, int16_t);
WARPBLOCK_GENREDU_ALLOP(fixed2u, uint16_t);
WARPBLOCK_GENREDU_ALLOP(fixed4, int32_t);
WARPBLOCK_GENREDU_ALLOP(fixed4u, uint32_t);
WARPBLOCK_GENREDU_ALLOP(fixed8, int64_t);
WARPBLOCK_GENREDU_ALLOP(fixed8u, uint64_t);

#define WARPBLOCK_GENREDU_ALLOP_F(_name, _dtype)                               \
  WARPBLOCK_GENREDU(_name, _dtype, add);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, sub);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, mul);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, div);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, min);                                       \
  WARPBLOCK_GENREDU(_name, _dtype, max);
WARPBLOCK_GENREDU_ALLOP_F(float4, float);
WARPBLOCK_GENREDU_ALLOP_F(float8, double);

//
// runtime support for array reduction
//

#define ARRAYATOMIC_GENOP(_name, _dtype, _op)                                  \
  EXTERN void __array_atomic_##_name##_##_op(                                  \
      kmp_Indent *id_ref, int32_t gtid, _dtype *lhs, _dtype *rhs, int64_t n) { \
    PRINT(LD_LOOP, "Reduction: thead %d\n", gtid);                             \
    for (int i = 0; i < n / sizeof(_dtype); i++) {                             \
      __kmpc_atomic_##_name##_##_op(id_ref, gtid, lhs + i, rhs[i]);            \
    }                                                                          \
  }                                                                            \
  EXTERN void __gpu_array_warpBlockRedu_##_name##_##_op(_dtype *ldata,         \
                                                        int64_t n) {           \
    for (int i = 0; i < n / sizeof(_dtype); i++) {                             \
      ldata[i] = __gpu_warpBlockRedu_##_name##_##_op(ldata[i]);                \
    }                                                                          \
  }

#define ARRAY_GEN_ALLOP_INTEGER(_name, _tname, _optype)                        \
  _name(_tname, _optype, add);                                                 \
  _name(_tname, _optype, sub);                                                 \
  _name(_tname, _optype, mul);                                                 \
  _name(_tname, _optype, div);                                                 \
  _name(_tname, _optype, min);                                                 \
  _name(_tname, _optype, max);                                                 \
  _name(_tname, _optype, andb);                                                \
  _name(_tname, _optype, orb);                                                 \
  _name(_tname, _optype, xor);                                                 \
  _name(_tname, _optype, shl);                                                 \
  _name(_tname, _optype, shr);                                                 \
  _name(_tname, _optype, andl);                                                \
  _name(_tname, _optype, orl);                                                 \
  _name(_tname, _optype, eqv);                                                 \
  _name(_tname, _optype, neqv);

#define ARRAY_GEN_ALLOP_FLOAT(_name, _tname, _optype)                          \
  _name(_tname, _optype, add);                                                 \
  _name(_tname, _optype, sub);                                                 \
  _name(_tname, _optype, mul);                                                 \
  _name(_tname, _optype, div);                                                 \
  _name(_tname, _optype, min);                                                 \
  _name(_tname, _optype, max);

ARRAY_GEN_ALLOP_INTEGER(ARRAYATOMIC_GENOP, fixed1, int8_t);
ARRAY_GEN_ALLOP_INTEGER(ARRAYATOMIC_GENOP, fixed2, int16_t);
ARRAY_GEN_ALLOP_INTEGER(ARRAYATOMIC_GENOP, fixed4, int32_t);
ARRAY_GEN_ALLOP_INTEGER(ARRAYATOMIC_GENOP, fixed8, int64_t);
ARRAY_GEN_ALLOP_FLOAT(ARRAYATOMIC_GENOP, float4, float);
ARRAY_GEN_ALLOP_FLOAT(ARRAYATOMIC_GENOP, float8, double);
